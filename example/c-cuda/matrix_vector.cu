#include "hip/hip_runtime.h"
#include <stdio.h>
// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

/**
 * Matrix vector multiplication (CUDA Kernel) on the device: c = A * b
 * wA is A's width (# of columns) (the same that b's large (or # of rows))
 */
__global__
void kernel(float *A, float *b, float *c, const int N, const int M){
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
  float sum=0;
  if(tid<N){
    for(int i=0; i<M; i++)
      sum += b[i]*A[(tid*M)+i];
    c[tid]=sum;
    }
}

// debuging functions
void init_array(float *a, const int N);
void init_mat(float *a, const int N, const int M);
void print_array(float *a, const int N, char *d);
void print_mat(float *a, const int N, const int M, char *d);

/**
  * Main function
  */
int main (int argc, char **argv) {


  printf("***********************************************************\n");
  printf("[Matrix Vector multiplication using CUDA]\n");
  printf(" Performs matrix multiplication c = A*b , where: \n");
  printf("     A is a NxM matrix\n"); 
  printf("     b is a Mx1 vector\n"); 
  printf("     c is a Nx1 vector\n");
  printf("***********************************************************\n\n");

  printf(" ........ Starting.......\n");


    //checkCmdLineFlag is a function provided by helper_functions.h (Cuda toolkit) 
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("     -wA=WidthA (M) -hA=HeightA (N) (A of size [NxM])\n");
        printf("     Note: Size of vector b must be the same than the width of matrix A ([Mx1])\n");

        exit(EXIT_SUCCESS);
    }



  int numDev = 0;   
  int devID = 0;
  hipError_t error;
  hipDeviceProp_t deviceProp;
  
  hipGetDeviceCount(&numDev);

  // Get CUDA device access from device id
  error = hipGetDevice(&devID);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

  // Get properties from CUDA device
  error = hipGetDeviceProperties(&deviceProp, devID);
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        // Print information about the graphic card (gpu) used to run this example
        printf("GPU Devices installed on the cluster: %d \n", numDev);

        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID,
            deviceProp.name, deviceProp.major, deviceProp.minor);
        printf("GPU Device num of multiprocessors:  %d \n", deviceProp.multiProcessorCount);
        printf("GPU Device num of thread per block: %d \n", 
            deviceProp.maxThreadsPerBlock);
        printf("GPU Device num of thread per multiprocessor: %d \n", 
            deviceProp.maxThreadsPerMultiProcessor);



    }


  float *A, *b, *c;
  float *dev_A, *dev_b, *dev_c;

  int N=3;
  int M=4;
  
  // Memory allocation using the typical C way
  A=(float*)malloc(sizeof(float)*N*M);
  b=(float*)malloc(sizeof(float)*M);
  c=(float*)malloc(sizeof(float)*N);

  // Memory allocation in the device (gpu)
  hipMalloc((void**)&dev_A, sizeof(float)*N*M);
  hipMalloc((void**)&dev_b, sizeof(float)*M);
  hipMalloc((void**)&dev_c, sizeof(float)*N);
  
  
  // Functions to initialize with some values matrix A, vector b and vector c
  init_mat(A, N, M);
  init_array(b, M);
  init_array(c, N);


  // Displaying values assigned to initial vectors and matrix
  printf("******** Initializing matrices data randomly ********\n");
  print_mat(A, N, M, "matrix (A) [NXM]");
  print_array(b, M, "in-vector (b) [Mx1]");
  print_array(c, N, "out-vector (c) [Nx1]");



  // Transfering memory from the cpu (host) to the gpu (device)
  hipMemcpy(dev_A, A, sizeof(float)*N*M, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(float)*M, hipMemcpyHostToDevice);


  // Kernel code. This portion of code is executed directly in the device (gpu)
  printf("\n\nRunning Kernel...\n\n");
  kernel<<<(N+1023)/1024, 1024>>>(dev_A, dev_b, dev_c, N, M);

  
  // Transfer the result from device memory to cpu memory
  hipMemcpy(c, dev_c, sizeof(float)*N, hipMemcpyDeviceToHost);

  // Cleaning memory from the device
  hipFree(dev_A);
  hipFree(dev_b);
  hipFree(dev_c);


  // Displaying the results
  printf("******** Output data ********\n");
  print_array(c, N, "out-vector (c) [Nx1]");

  printf(" ........ Finished.......\n");


  return 0;

};

// Random initialization of a vector
void init_array(float *a, const int N) {
  int i;
  for(i=0; i<N; i++)
    a[i] = rand() % 4 + 1;
}
// Random initialization of a matrix
void init_mat(float *a, const int N, const int M) {
  int i, j;
  for(i=0; i<N; i++)
    for(j=0; j<M; j++)
      a[i*M+j] = rand() % 4 + 1;
}
// Display a vector values
void print_array(float *a, const int N, char *d) {
 int i;
 for(i=0; i<N; i++)
   printf("\n%s[%d]: %f",d, i, a[i]);
 printf("\n");
}
// Display a matrix values
void print_mat(float *a, const int N, const int M, char *d) {
  int i, j;
  for(i=0; i<N; i++){
    printf("\n%s[%d]:", d, i);
      for (j=0; j<M; j++)
        printf("\t%6.4f", a[i*M+j]);
   }
   printf("\n");
}
